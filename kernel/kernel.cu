#include "hip/hip_runtime.h"
#include <kernel.cuh>
#include <chrono>
#include <iostream>

using namespace bodies;

const double k = 8.99e9;


hipDeviceProp_t getDetails(int deviceId)
{
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceId);
    return props;
}


void PrintProps(hipDeviceProp_t props) {
    std::cout << "Device: " << props.name << std::endl;
    std::cout << "\tSMs: " << props.multiProcessorCount << std::endl;
    std::cout << "\tMax Blocks Per SM: " << props.maxBlocksPerMultiProcessor << std::endl;
    std::cout << "\tMax Threads Per SM: " << props.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "\tMax Threads Per Block: " << props.maxThreadsPerBlock << std::endl;
    std::cout << "\tWarpSize: " << props.warpSize << std::endl;
    

}


__device__ double getMagnitudeDistance(bodies::pVec p, bodies::pVec p2) {

    double dx = p.x - p2.x;
    double dy = p.y - p2.y;
    double dz = p.z - p2.z;

    double magnitude = sqrt((dx * dx) + (dy * dy) + (dz * dz));  // |x| = sqrt(x^2 + y^2 + z^2)
    return magnitude;
}

__global__ void calculatePotentials(bodies::pVec *point, bodies::body *particles, double *Ve, int N, int Np) {
    
    int index =  blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    double total;

    for (int i = index; i < Np; i+= stride) {
        total = 0.0;
        
        for (int j = 0; j < N; j++) {
            double r = getMagnitudeDistance(point[index], particles[i].position);
            total += k * (particles[i].charge / r);
        }
        
        Ve[i] = total;
    }
    
}


double hostGetMagnitudeDistance(bodies::pVec p, bodies::pVec p2) {
    double dx = p.x - p2.x;
    double dy = p.y - p2.y;
    double dz = p.z - p2.z;

    double magnitude = sqrt((dx * dx) + (dy * dy) + (dz * dz));  // |x| = sqrt(x^2 + y^2 + z^2)
    return magnitude;
}

double* kernel::CPUlaunch(bodies::pVec *points, bodies::body *particles, int N, int Np) {

    double r;
    double total = 0.0f;

    auto Ve = new double[Np];

    for (int j = 0; j < Np; j++) {
        total = 0.0;
        for (int i = 0; i < N; i++) {
            r = hostGetMagnitudeDistance(points[j], particles->position);
            total += k * (particles[i].charge / r);
            
        }
        Ve[j] = total;
    }


    /**
    printf("CPU Result = \n");
    for (int i = 0; i < Np; i++) {
        printf("%ld,", Ve[i]);
    }
    printf("\n");
    **/ 
    return Ve;
}

double *kernel::launch(bodies::pVec *points, bodies::body *particles, int N, int Np) {   

    int deviceID;
    hipGetDevice(&deviceID);
    hipDeviceProp_t props = getDetails(deviceID);

    size_t size = sizeof(bodies::body) * N;

    hipError_t stepError;
    hipError_t asyncError;

    bodies::body *d_particles;
    double *d_Ve;
    double *Ve;
    bodies::pVec *d_points;
    
    hipMalloc((void **)&d_points, sizeof(bodies::pVec) * Np);
    hipMalloc((void **)&d_Ve, sizeof(double) * Np);
    hipMalloc((void **)&d_particles, size);
    Ve = new double[Np];

    hipStream_t particleStream, pointStream;
    hipStreamCreate(&particleStream);
    hipStreamCreate(&pointStream);

    stepError = hipGetLastError();
    if (stepError != hipSuccess){std::cout << "! (Alloc) STEP ERROR: " << hipGetErrorString(stepError) << std::endl;}
    asyncError = hipDeviceSynchronize();
    if(asyncError != hipSuccess){std::cout << "! (Alloc) ASYNC ERROR" << std::endl;}

    hipMemcpyAsync(d_particles, particles, size, hipMemcpyHostToDevice, particleStream);
    hipMemcpyAsync(d_points, points, size, hipMemcpyHostToDevice, pointStream);

    int threadsPerBlock = 512; // (16 warps worth)
    int blocks = props.multiProcessorCount * props.maxBlocksPerMultiProcessor;
    //std::cout << "Kernel configuration = <" << threadsPerBlock << "," << blocks << ">\n";
    
    calculatePotentials<<<threadsPerBlock, blocks>>>(d_points, d_particles, d_Ve, N, Np);
    
    // copy back array of N potentials
    hipMemcpy(Ve, d_Ve, sizeof(double) * Np, hipMemcpyDeviceToHost);
    // sum potentials

    stepError = hipGetLastError();
    //if (stepError != hipSuccess){std::cout << "! STEP ERROR: " << hipGetErrorString(stepError) << std::endl;}
    asyncError = hipDeviceSynchronize();
    if(asyncError != hipSuccess){std::cout << "! ASYNC ERROR" << std::endl;}

    hipFree(d_particles); hipFree(d_Ve); hipFree(d_points);

    /**
    printf("GPU Result = \n");
    for (int i = 0; i < Np; i++) {
        printf("%ld,", Ve[i]);
    }
    printf("\n");
    **/

    return Ve;
}